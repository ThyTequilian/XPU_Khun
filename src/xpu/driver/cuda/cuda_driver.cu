#include <xpu/host.h>

#include <iostream>

class cuda_driver : public xpu::driver_interface {

public:
    xpu::error setup() override{
        std::cout << "xpu: CUDA SETUP" << std::endl;

        xpu::error err;
        hipDeviceProp_t props;
        err = hipGetDeviceProperties(&props, 0);
        if (err != 0) {
            return err;
        }
    
        std::cout << "xpu: selected cuda device " << props.name << "(" << props.major << props.minor << ")" << std::endl;
    
        err = hipSetDevice(0);
        if (err != 0) {
            return err;
        }
        err = hipDeviceSynchronize();
    
        return err;
    }

    xpu::error device_malloc(void **ptr, size_t bytes) override {
        size_t free, total;
        hipMemGetInfo(&free, &total);
        std::cout << "cuda driver: free memory = " << free << "; total = " << total << std::endl;
        std::cout << "cuda driver: allocating " << bytes << " bytes" << std::endl;
        return hipMalloc(ptr, bytes);
    }

    xpu::error free(void *ptr) override {
        return hipFree(ptr);
    }

    xpu::error memcpy(void *dst, const void *src, size_t bytes) override {
        xpu::error err = hipMemcpy(dst, src, bytes, hipMemcpyDefault);
        hipDeviceSynchronize();
        return err;
    }

    xpu::error memset(void *dst, int ch, size_t bytes) override {
        return hipMemset(dst, ch, bytes);
    }

};

extern "C" xpu::driver_interface *create() {
    return new cuda_driver{};
}

extern "C" void destroy(xpu::driver_interface *b) {
    delete b;
}