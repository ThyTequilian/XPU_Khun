#include "../../detail/driver_interface.h"

#include <iostream>

#define CU_DEVICE 0

using xpu::detail::error;

class cuda_driver : public xpu::detail::driver_interface {

public:
    error setup() override {
        std::cout << "xpu: CUDA SETUP" << std::endl;

        error err;
        hipDeviceProp_t props;
        err = hipGetDeviceProperties(&props, CU_DEVICE);
        if (err != 0) {
            return err;
        }

        std::cout << "xpu: selected cuda device " << props.name << "(" << props.major << props.minor << ")" << std::endl;

        err = hipSetDevice(CU_DEVICE);
        if (err != 0) {
            return err;
        }
        err = hipDeviceSynchronize();

        return err;
    }

    error device_malloc(void **ptr, size_t bytes) override {
        size_t free, total;
        hipMemGetInfo(&free, &total);
        std::cout << "cuda driver: free memory = " << free << "; total = " << total << std::endl;
        std::cout << "cuda driver: allocating " << bytes << " bytes" << std::endl;
        return hipMalloc(ptr, bytes);
    }

    error free(void *ptr) override {
        return hipFree(ptr);
    }

    error memcpy(void *dst, const void *src, size_t bytes) override {
        error err = hipMemcpy(dst, src, bytes, hipMemcpyDefault);
        hipDeviceSynchronize();
        return err;
    }

    error memset(void *dst, int ch, size_t bytes) override {
        return hipMemset(dst, ch, bytes);
    }

};

extern "C" xpu::detail::driver_interface *create() {
    return new cuda_driver{};
}

extern "C" void destroy(xpu::detail::driver_interface *b) {
    delete b;
}
