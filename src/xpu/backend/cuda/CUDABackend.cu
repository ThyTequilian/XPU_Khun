#include "CUDABackend.h"

GPUError CUDABackend::setup() {
    hipSetDevice(0);
    hipDeviceSynchronize();
    return 0;
}

GPUError CUDABackend::deviceMalloc(void **ptr, size_t bytes) {
    return hipMalloc(ptr, bytes);
}

GPUError CUDABackend::free(void *ptr) {
    return hipFree(ptr);
}

GPUError CUDABackend::memcpy(void *dst, const void *src, size_t bytes) {
    return hipMemcpy(dst, src, bytes, hipMemcpyDefault);
}

extern "C" GPUBackend *create() {
    return new CUDABackend{};
}

extern "C" void destroy(GPUBackend *b) {
    delete b;
}